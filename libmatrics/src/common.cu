#include "internal.h"

__export void mtNewStream(hipStream_t *pStream, int *pCode) {
    *pStream = nullptr;
    hipStream_t stream;
    if (hipSuccess != (*pCode = hipStreamCreate(&stream))) return;
    *pStream = stream;
}

__export void mtStreamDestroy(hipStream_t stream) {
    hipStreamDestroy(stream);
}

void newBuffer(uint32 sz, buffer *pBuf, int *pCode) {
    *pBuf = nullptr;
    buffer buf;
    if (hipSuccess != (*pCode = hipMallocManaged(&buf, sz))) return;
    *pBuf = buf;
}

__export void mtNewBuffer(uint32 sz, buffer *pBuf, int *pCode) {
    newBuffer(sz, pBuf, pCode);
}

__export void mtBufferDestroy(buffer buf) {
    hipFree(buf);
}
