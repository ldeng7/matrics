#include "internal.h"

__export void mtNewStream(hipStream_t *pStream, int *pCode) {
    *pStream = nullptr;
    hipStream_t stream;
    if (hipSuccess != (*pCode = hipStreamCreate(&stream))) return;
    *pStream = stream;
}

__export void mtStreamDestroy(hipStream_t stream) {
    hipStreamDestroy(stream);
}

__export void mtNewTensor(size_t nx, size_t ny, size_t nz, size_t nw,
        MtTensor **ppTen, float **ppBuf, int *pCode) {
    *ppTen = nullptr;
    MtTensor *pTen = (MtTensor *)(malloc(sizeof(MtTensor)));
    if (nullptr == pTen) {
        *pCode = cudaErrorApiFailureBase;
        return;
    }
    pTen->nx = nx;
    pTen->ny = ny;
    pTen->nz = nz;
    pTen->nw = nw;
    size_t sz = sizeof(float) * nx * ny * nz * nw;
    if (hipSuccess != (*pCode = hipMallocManaged(&(pTen->p), sz))) {
        free(pTen);
        return;
    }
    *ppTen = pTen;
    *ppBuf = pTen->p;
}

__export void mtTensorDestroy(MtTensor *pTen) {
    if (nullptr != pTen->p) hipFree(pTen->p);
    free(pTen);
}

MtBuffer mtNewBuffer(size_t n, int *pCode) {
    MtBuffer buf;
    size_t sz = sizeof(float) * n;
    *pCode = hipMallocManaged(&buf, sz);
    return buf;
}

__export void mtBufferDestroy(MtBuffer buf) {
    hipFree(buf);
}
