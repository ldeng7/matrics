#include "internal.h"

__export void mtNewStream(hipStream_t *pStream, int *pCode) {
    *pStream = nullptr;
    hipStream_t stream;
    if (hipSuccess != (*pCode = hipStreamCreate(&stream))) return;
    *pStream = stream;
}

__export void mtStreamDestroy(hipStream_t stream) {
    hipStreamDestroy(stream);
}

MtBuffer mtNewBuffer(size_t n, int *pCode) {
    MtBuffer buf;
    size_t sz = sizeof(float) * n;
    *pCode = hipMallocManaged(&buf, sz);
    return buf;
}

__export void mtBufferDestroy(MtBuffer buf) {
    hipFree(buf);
}
